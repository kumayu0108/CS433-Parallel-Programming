#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
namespace cg = cooperative_groups;

#define TOL 1e-5
#define ITER_LIMIT 1000
#define TILE_X 16
#define TILE_Y 16
#define THREADS_PER_BLOCK 1024
#define TPBX TILE_X
#define TPBY TILE_Y

__managed__ float diff = 0;
__managed__ int done = 0;
__managed__ int iter = 0;

__global__ void init_kernel (float *a, int n, int t, int span, hiprandState *states)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(id, id, 0, &states[id]);
    for (int i = id * span; i < (id + 1) * span; i++)
    {
        a[i] = hiprand_uniform(&states[id]);
    }
    if (threadIdx.x == blockDim.x - 1 && blockIdx.x == gridDim.x - 1)
    {
        for (int i = (id + 1) * span; i < (n + 2) * (n + 2); i++)
        {
            a[i] = hiprand_uniform(&states[id]);
        }
    }

}

__global__ void solver(float *a, int n, int dim_span_x, int dim_span_y)
{
    float local_diff = 0.0, temp;
    int x_id, y_id;
    int ij, ipj, ijm, imj, ijp;
    cg::grid_group grid = cg::this_grid();
    while (!done)
    {
        __shared__ float atile[TILE_X + 2][TILE_Y + 2];
        local_diff = 0;
        x_id = blockIdx.x * blockDim.x + threadIdx.x;
        y_id = blockIdx.y * blockDim.y + threadIdx.y;
        if(x_id == 0 && y_id == 0)
        {
            diff = 0;
        }
        grid.sync();

        for (int m = 0; m < dim_span_x; m++)
        {
            for (int n = 0; n < dim_span_y; n++)
            {
                int tidx = threadIdx.x;
                int tidy = threadIdx.y;

                int i = (blockIdx.x * dim_span_x + m) * TILE_X + threadIdx.x;
                int j = (blockIdx.y * dim_span_y + n) * TILE_Y + threadIdx.y;
                // printf("%d %d\n", i, j);

                ij = (i + 1) * (n + 2) + (j + 1);
                ipj = (i + 2) * (n + 2) + (j + 1);
                imj = (i) * (n + 2) + (j + 1);
                ijp = (i + 1) * (n + 2) + (j + 2);
                ijm = (i + 1) * (n + 2) + (j);
                __syncthreads();
                atile[tidx + 1][tidy + 1] = a[ij];
                atile[tidx][tidy + 1] = a[imj];
                atile[tidx + 2][tidy + 1] = a[ipj];
                atile[tidx + 1][tidy] = a[ijm];
                atile[tidx + 1][tidy + 2] = a[ijp ];
                __syncthreads();

                temp = a[ij];
                a[ij] = 0.2 * (atile[tidx + 1][tidy + 1] + atile[tidx][tidy + 1] + atile[tidx + 2][tidy + 1] + atile[tidx + 1][tidy] + atile[tidx + 1][tidy + 2]);
                local_diff+=fabs(a[ij] - temp);
                __syncthreads();
            }
        }
        atomicAdd(&diff, local_diff);
        if(x_id == 0 && y_id == 0)
        {
            iter++;
        }
        grid.sync();
        if((diff / (n * n)) < TOL || (iter == ITER_LIMIT))
        {
            if(x_id == 0 && y_id == 0)
                printf("Thread {0, 0} : Iterations %d  Error %.7f\n", iter, diff/(n*n));
            done = 1;
        }
        grid.sync();
    }
}

int main (int argc, char *argv[])
{

	float *a;
    int n, t, init_span, total_span, pt_span_x, pt_span_y;
    int num_thread_blocks_x, num_thread_blocks_y;
    int num_threads_per_block_x = 1, num_threads_per_block_y = 1;
    struct timeval tv0, tv2;
    struct timezone tz0, tz2;

    n = atoi(argv[1]);
    t = atoi(argv[2]);
    assert((t  & (t - 1)) == 0);
    if(n * n < t)
    {
        t = n * n;
        printf("more threads than number of elts, reducing them; nthreads now %d\n", t);
    }
    int log_span = (int)log2((float)t);
    int half = log_span / 2;
    int left = log_span - half;
    // everything is a square, so calculated in one dimension
    num_thread_blocks_x = (1<<left) / TILE_X;
    num_thread_blocks_y = (1<<half) / TILE_Y;
    num_threads_per_block_x = TILE_X;
    num_threads_per_block_y = TILE_Y;
    total_span = (n * n) / t;
    half = ((int)log2((float)total_span)) / 2;
    left = ((int)log2((float)total_span)) - half;
    pt_span_x = (1<<half);
    pt_span_y = (1<<left);

    assert(num_thread_blocks_x * num_threads_per_block_x * num_thread_blocks_y * num_threads_per_block_y == t);

	hipMallocManaged((void**)&a, sizeof(float) * (n + 2) * (n + 2));

    int device = -1;
    hipGetDevice(&device);
    hipMemAdvise(a, sizeof(float) * (n + 2) * (n + 2), hipMemAdviseSetPreferredLocation, device);

    if((n + 2) * (n + 2) >= THREADS_PER_BLOCK)
    {
        int num_blocks = ((n + 2) * (n + 2)) / THREADS_PER_BLOCK;
        int tot_threads = num_blocks * THREADS_PER_BLOCK;
        init_span = ((n + 2) * (n + 2))/ tot_threads;
        hiprandState *dev_random;
        hipMalloc((void**)&dev_random, tot_threads * sizeof(hiprandState));
        init_kernel<<<((n + 2) * (n + 2)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(a, n, t, init_span, dev_random);
    }
    else
    {
        init_span = 1;
        int tot_threads = (n + 2) * (n + 2);
        hiprandState *dev_random;
        hipMalloc((void**)&dev_random, tot_threads * sizeof(hiprandState));
        init_kernel<<<1, (n + 2) * (n + 2)>>>(a, n, t, init_span, dev_random);
    }
    // printf("choosing init span: %d, normal span: %d\n\n", init_span, total_span);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
	hipDeviceSynchronize();

    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, device);
    if(!supportsCoopLaunch)
    {
        printf("\nno cooperative groups launch, exiting\n");
        exit(1);
    }

    void* kernelArgs[] = {(void*)&a, (void*)&n, (void*)&pt_span_x, (void*)&pt_span_y};
    dim3 dimBlock(num_threads_per_block_x, num_threads_per_block_y);
    dim3 dimGrid(num_thread_blocks_x, num_thread_blocks_y);

    gettimeofday(&tv0, &tz0);
    hipLaunchCooperativeKernel((void*)solver, dimGrid, dimBlock, kernelArgs);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
	hipDeviceSynchronize();
    gettimeofday(&tv2, &tz2);

    printf("time: %ld microseconds\n", (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
	return 0;
}
