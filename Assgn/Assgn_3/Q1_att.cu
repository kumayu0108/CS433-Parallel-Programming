#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
namespace cg = cooperative_groups;
 
#define TOL 1e-5
#define ITER_LIMIT 1000
#define THREADS_PER_BLOCK 1024
 
__managed__ float diff = 0;
__managed__ int done = 0;
__managed__ int iter = 0;
 
__global__ void init_kernel (float *a, int n, int t, int span, hiprandState *states)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(id, id, 0, &states[id]);
    for (int i = id * span; i < (id + 1) * span; i++)
    {
        a[i] = hiprand_uniform(&states[id]);
    }
    if (threadIdx.x == blockDim.x - 1 && blockIdx.x == gridDim.x - 1)
    {
        printf("in last thread, %d of %d threads, %d of %d blocks\n", threadIdx.x + 1, blockDim.x, blockIdx.x + 1, gridDim.x);
        printf("filling from %d to %d\n", (id + 1) * span, (n + 2) * (n + 2) - 1);
        for (int i = (id + 1) * span; i < (n + 2) * (n + 2); i++)
        {
            a[i] = hiprand_uniform(&states[id]);
        }
    }
 
}
 
__global__ void solver(float *a, int n, int x_tile, int y_tile)
{
    float local_diff = 0.0, temp;
    __shared__ float atile[68][68];
    int x_id, y_id;
    int ij, ipj, ijm, imj, ijp;
    cg::grid_group grid = cg::this_grid();
    while (!done)
    {
        local_diff = 0;
        x_id = blockIdx.x * blockDim.x + threadIdx.x;
        y_id = blockIdx.y * blockDim.y + threadIdx.y;
        if(x_id == 0 && y_id == 0)
        {
            diff = 0;
        }
        grid.sync();
 
        for (int i = x_id * x_tile; i < (x_id + 1) * x_tile; i++)
        {
            for (int j = y_id * y_tile; j < (y_id + 1) * y_tile; j++)
            {
                int tidx = i % x_tile;
                int tidy = j % y_tile;
                ij = (i + 1) * (n + 2) + (j + 1);
                ipj = (i + 2) * (n + 2) + (j + 1);
                imj = (i) * (n + 2) + (j + 1);
                ijp = (i + 1) * (n + 2) + (j + 2);
                ijm = (i + 1) * (n + 2) + (j);
                atile[tidx + 1][tidy + 1] = a[ij];
                atile[tidx + 1][tidy + 2] = a[ijp];
                atile[tidx + 2][tidy + 1] = a[ipj];
                atile[tidx][tidy + 1] = a[imj];
                atile[tidx + 1][tidy] = a[ijm];
                temp = a[ij];
                a[ij] = 0.2 * (atile[tidx + 1][tidy + 1] + atile[tidx + 1][tidy + 2] + atile[tidx + 2][tidy + 1] + atile[tidx][tidy + 1] + atile[tidx + 1][tidy]);
                local_diff+=fabs(a[ij] - temp);
                // printf("[iter: %d] %d, %d, %d, %d, %d %f\n", iter, ij, ipj, imj, ijp, ijm, local_diff);
            }
        }
        atomicAdd(&diff, local_diff);
        if(x_id == 0 && y_id == 0)
        {
            iter++;
            // printf("[iter: %5d] diff: %6f, local: %6f\n", iter, diff/(n * n), local_diff);
        }
        grid.sync();
        if((diff / (n * n)) < TOL || (iter == ITER_LIMIT))
        {
            done = 1;
        }
        grid.sync();
    }
}
 
int main (int argc, char *argv[])
{
 
    float *a;
    int n, t, left, init_span, total_span;
    int x_tile, y_tile;
    int num_thread_blocks_x, num_thread_blocks_y;
    int num_threads_per_block_x = 1, num_threads_per_block_y = 1;
    struct timeval tv0, tv2;
    struct timezone tz0, tz2;
 
 
 
    n = atoi(argv[1]);
    t = atoi(argv[2]);
    assert((t  & (t - 1)) == 0);
    if(n * n < t)
    {
        t = n * n;
        printf("more threads than number of elts, reducing them; nthreads now %d\n", t);
    }
    total_span = ((n) * (n)) / t;
    int log_t_2 = (int)log2((float)t);
    int log_span_2 = (int)log2((float)total_span);
    if(log_t_2 < 10) // less than THREADS_PER_BLOCK
    {
        int half = log_t_2 / 2;
        left = log_t_2 - half;
        num_threads_per_block_x = (1<<left);
        num_threads_per_block_y = (1<<half);
    }
    else
    {
        num_threads_per_block_x = 32;
        num_threads_per_block_y = 32;
        left = log_t_2 - 10;
    }
 
    int half = log_span_2 / 2;
    left = log_span_2 - half;
    x_tile = (1<<half);
    y_tile = (1<<left);
 
    num_thread_blocks_x = n / (x_tile * num_threads_per_block_x);
    num_thread_blocks_y = n / (y_tile * num_threads_per_block_y);
 
    printf("choosing gridDims: %d, %d\n", num_thread_blocks_x, num_thread_blocks_y);
    printf("choosing blockDims: %d, %d\n", num_threads_per_block_x, num_threads_per_block_y);
    printf("choosing tiles: %d, %d\n", x_tile, y_tile);
 
    assert(num_thread_blocks_x * num_threads_per_block_x * num_thread_blocks_y * num_threads_per_block_y == t);
    assert(x_tile * y_tile == total_span);
    assert(x_tile * num_threads_per_block_x * num_thread_blocks_x == n);
    assert(y_tile * num_threads_per_block_y * num_thread_blocks_y == n);
 
 
 
    hipMallocManaged((void**)&a, sizeof(float) * (n + 2) * (n + 2));
 
    int device = -1;
    hipGetDevice(&device);
    hipMemAdvise(a, sizeof(float) * (n + 2) * (n + 2), hipMemAdviseSetPreferredLocation, device);
 
    if((n + 2) * (n + 2) >= THREADS_PER_BLOCK)
    {
        int num_blocks = ((n + 2) * (n + 2)) / THREADS_PER_BLOCK;
        int tot_threads = num_blocks * THREADS_PER_BLOCK;
        init_span = ((n + 2) * (n + 2))/ tot_threads;
        hiprandState *dev_random;
        hipMalloc((void**)&dev_random, tot_threads * sizeof(hiprandState));
        init_kernel<<<((n + 2) * (n + 2)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(a, n, t, init_span, dev_random);
    }
    else
    {
        init_span = 1;
        int tot_threads = (n + 2) * (n + 2);
        hiprandState *dev_random;
        hipMalloc((void**)&dev_random, tot_threads * sizeof(hiprandState));
        init_kernel<<<1, (n + 2) * (n + 2)>>>(a, n, t, init_span, dev_random);
    }
    // printf("choosing init span: %d, normal span: %d\n\n", init_span, total_span);
 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
 
    // for(int i = 0; i < 32; i++)
    // {
    //     printf("%f ", a[i]);
    // }
    // printf("\n\n");
 
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, device);
    if(!supportsCoopLaunch)
    {
        printf("\nno cooperative groups launch, exiting\n");
        exit(1);
    }
 
    void* kernelArgs[] = {(void*)&a, (void*)&n, (void*)&x_tile, (void*)&y_tile};
    dim3 dimBlock(num_threads_per_block_x, num_threads_per_block_y);
    dim3 dimGrid(num_thread_blocks_x, num_thread_blocks_y);
 
    gettimeofday(&tv0, &tz0);
    hipLaunchCooperativeKernel((void*)solver, dimGrid, dimBlock, kernelArgs);
 
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    gettimeofday(&tv2, &tz2);
 
    printf("time: %ld microseconds\n", (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
    return 0;
}