#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SIZE (1<<28)

__global__ void vadd_kernel (float *a, float *b, float *c, int span)
{
	int i;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	
	for (i=span*id; i<span*(id+1); i++) {
		c[i] = a[i] + b[i];
	}
}	

int main (int argc, char *argv[])
{
	int i;
	float *a, *b, *c;
	int nthreads;
	struct timeval tv0, tv2;
	struct timezone tz0, tz2;

	if (argc != 2) {
		printf ("Need number of threads.\n");
		exit(1);
	}
	nthreads = atoi(argv[1]);
	assert((nthreads & (nthreads - 1)) == 0);

	hipMallocManaged((void**)&a, sizeof(float)*SIZE);
	for (i=0; i<SIZE; i++) a[i] = 1;

	hipMallocManaged((void**)&b, sizeof(float)*SIZE);
        for (i=0; i<SIZE; i++) b[i] = 1.5;

	int device = -1;
        hipGetDevice(&device);

	hipMemAdvise(a, sizeof(float)*SIZE, hipMemAdviseSetReadMostly, 0);
	hipMemAdvise(b, sizeof(float)*SIZE, hipMemAdviseSetReadMostly, 0);

        hipMemPrefetchAsync(a, sizeof(float)*SIZE, device, NULL);
        hipMemPrefetchAsync(b, sizeof(float)*SIZE, device, NULL);

	hipMallocManaged((void**)&c, sizeof(float)*SIZE);
	hipMemAdvise(c, sizeof(float)*SIZE, hipMemAdviseSetPreferredLocation, device);

	gettimeofday(&tv0, &tz0);

	if (nthreads < 16) {
		vadd_kernel<<<1, nthreads>>>(a, b, c, SIZE/nthreads);
	}
	else {
		vadd_kernel<<<nthreads/8, 8>>>(a, b, c, SIZE/nthreads);
	}
	hipDeviceSynchronize();

	gettimeofday(&tv2, &tz2);
	
	printf("Random element: %lf, time: %ld microseconds\n", c[random() % SIZE], (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
	return 0;
}
