#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define N (1ULL << 34)

#define START_X 0
#define END_X 1.0

__managed__ float area;

__device__ float compute_f (float x)
{
	return 1.0/(1.0 + x*x);
}

__global__ void area_kernel (float a, float b, unsigned long long num_intervals_per_thread)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	float sub_interval = (b - a)/N;
	float x = a + id*sub_interval*num_intervals_per_thread;
	unsigned long long i;
	float local_area = 0;
	for (i=0; i<num_intervals_per_thread; i++) {
		local_area += (compute_f (x) + compute_f (x+sub_interval));
		x += sub_interval;
	}
	unsigned mask = 0xffffffff;
	for (i=warpSize/2; i>0; i=i/2) local_area += __shfl_xor_sync(mask, local_area, i);
	if (threadIdx.x == 0) {
		atomicAdd(&area, 0.5*local_area);
	}
}	

int main (int argc, char *argv[])
{
	unsigned long long nthreads;
	struct timeval tv0, tv2;
	struct timezone tz0, tz2;

	if (argc != 2) {
		printf ("Need number of threads.\n");
		exit(1);
	}
	nthreads = atoll(argv[1]);
	assert((nthreads & (nthreads - 1)) == 0);
	if (nthreads > N) nthreads = N;

	area = 0;
	int device = -1;
        hipGetDevice(&device);
        hipMemAdvise(&area, sizeof(float), hipMemAdviseSetPreferredLocation, device);

	gettimeofday(&tv0, &tz0);

	if (nthreads < 32) {
		area_kernel<<<1, nthreads>>>(START_X, END_X, N/nthreads);
	}
	else {
		area_kernel<<<nthreads/32, 32>>>(START_X, END_X, N/nthreads);
	}
	hipDeviceSynchronize();

	area = (area*(END_X-START_X))/N;

	gettimeofday(&tv2, &tz2);
	
	printf("Area: %.20f, time: %ld microseconds\n", area, (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
	return 0;
}
