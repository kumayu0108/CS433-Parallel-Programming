#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define THREADS_PER_BLOCK 1024
#define N (1 << 25)

__global__ void init_kernel (int *a)
{
	int i;

	for (i=threadIdx.x*(N/THREADS_PER_BLOCK); i<(threadIdx.x+1)*(N/THREADS_PER_BLOCK); i++) a[i] = N - i;
}

__global__ void bitonic_sort_kernel (int *a)
{
	unsigned bf_sz, stage, index1, index2, lower, pair;
	int init_log2stage, log2stage, temp;
	
	for (bf_sz = 2, init_log2stage = 0; bf_sz <= N; bf_sz = bf_sz*2, init_log2stage++) {
		for (stage = bf_sz/2, log2stage = init_log2stage; stage > 0; stage = stage/2, log2stage--) {
			for (pair=threadIdx.x*(N/(2*THREADS_PER_BLOCK)); pair<(threadIdx.x+1)*(N/(2*THREADS_PER_BLOCK)); pair++) {
				index1 = (pair >> log2stage) << 1;
				index2 = index1 | 1;
				lower = pair & ((1 << log2stage) - 1);
				index1 = (index1 << log2stage) | lower;
				index2 = (index2 << log2stage) | lower;
				if ((bf_sz & index1) == 0) {
					if (a[index1] > a[index2]) {
						temp = a[index1];
						a[index1] = a[index2];
						a[index2] = temp;
					}
				}
				else {
					if (a[index1] < a[index2]) {
                                        	temp = a[index1];
                                        	a[index1] = a[index2];
                                        	a[index2] = temp;
                                	}
				}
			}
			__syncthreads();
		}
	}
}	

int main (int argc, char *argv[])
{
	int i;
	int *a;
	struct timeval tv0, tv1;
	struct timezone tz0, tz1;

	hipMallocManaged((void**)&a, sizeof(int)*N);

	init_kernel<<<1, THREADS_PER_BLOCK>>>(a);

	hipDeviceSynchronize();

	gettimeofday(&tv0, &tz0);

	bitonic_sort_kernel<<<1, THREADS_PER_BLOCK>>>(a);

	hipDeviceSynchronize();

	gettimeofday(&tv1, &tz1);

	for (i=0; i<N-1; i++) {
		if (a[i] > a[i+1]) printf("Error at position %d, a[%d] = %d, a[%d] = %d\n", i, i, a[i], i+1, a[i+1]);
	}
	
	printf("Time: %ld microseconds\n", (tv1.tv_sec-tv0.tv_sec)*1000000+(tv1.tv_usec-tv0.tv_usec));
	return 0;
}
