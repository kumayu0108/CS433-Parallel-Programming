#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SIZE (1<<30)

__global__ void reduction_kernel (float *a, int span, double *target)
{
	int i;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	target[id] = 0;
	
	for (i=span*id; i<span*(id+1); i++) {
		target[id] += (a[i]*a[i]);
	}
}	

int main (int argc, char *argv[])
{
	int i;
	float *a, *gpu_a;
	int nthreads;
	double sum = 0;
	double *private_sum, *gpu_private_sum;
	struct timeval tv0, tv1, tv2;
	struct timezone tz0, tz1, tz2;

	if (argc != 2) {
		printf ("Need number of threads.\n");
		exit(1);
	}
	nthreads = atoi(argv[1]);
	private_sum = (double*)malloc(nthreads*sizeof(double));

	a = (float*)malloc(sizeof(float)*SIZE);
	assert(a != NULL);
	for (i=0; i<SIZE; i++) a[i] = 1;

	hipMalloc((void**)&gpu_a, sizeof(float)*SIZE);
	hipMalloc((void**)&gpu_private_sum, sizeof(double)*nthreads);

	gettimeofday(&tv0, &tz0);
	hipMemcpy(gpu_a, a, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	gettimeofday(&tv1, &tz1);

	if (nthreads < 16) {
		reduction_kernel<<<1, nthreads>>>(gpu_a, SIZE/nthreads, gpu_private_sum);
	}
	else {
		reduction_kernel<<<nthreads/16, 16>>>(gpu_a, SIZE/nthreads, gpu_private_sum);
	}
        hipMemcpy(private_sum, gpu_private_sum, sizeof(double)*nthreads, hipMemcpyDeviceToHost);

	for (i=0; i<nthreads; i++) sum += private_sum[i];

	gettimeofday(&tv2, &tz2);
	
	printf("SUM: %lf, time: %ld microseconds, copy time: %ld microseconds, compute time: %ld microseconds\n", sum, (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec), (tv1.tv_sec-tv0.tv_sec)*1000000+(tv1.tv_usec-tv0.tv_usec), (tv2.tv_sec-tv1.tv_sec)*1000000+(tv2.tv_usec-tv1.tv_usec));
	return 0;
}